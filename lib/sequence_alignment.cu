#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include "kernels/sequence_alignment_kernel.cuh"
#include "kernels/sequence_distance_kernel.cuh"
#include "wfa_types.h"
#include "utils/cuda_utils.cuh"
#include "utils/logger.h"
#include "sequence_alignment.cuh"

void allocate_offloaded_bt_d (wfa_backtrace_t** bt_offloaded_d,
                              const int max_steps,
                              const int num_blocks,
                              const size_t num_alignments) {
    size_t bt_offloaded_size = BT_OFFLOADED_ELEMENTS(max_steps);
    size_t max_addressable_elements = 1L << (sizeof(bt_prev_t) * 8);

    if (bt_offloaded_size >= max_addressable_elements) {
        LOG_ERROR("Trying to allocate more backtrace elements than the ones"
                  " that we can address.")
        exit(-1);
    }

    bt_offloaded_size *= num_blocks;

    // Add the results array
    size_t bt_offloaded_results_size = BT_OFFLOADED_RESULT_ELEMENTS(max_steps)
                                       * num_alignments;

    LOG_DEBUG("Allocating %.2f MiB to store backtraces of %zu alignments using %d blocks.",
              (float)((bt_offloaded_size + bt_offloaded_results_size) * sizeof(wfa_backtrace_t)) / (1 << 20),
              num_alignments, num_blocks)

    hipMalloc(bt_offloaded_d,
               (bt_offloaded_size + bt_offloaded_results_size) * sizeof(wfa_backtrace_t));
    CUDA_CHECK_ERR
}

void reset_offloaded_bt_d (wfa_backtrace_t* bt_offloaded_d,
                              const int max_steps,
                              const int num_blocks,
                              const size_t num_alignments,
                              hipStream_t stream) {
    size_t bt_offloaded_size = BT_OFFLOADED_ELEMENTS(max_steps);

    bt_offloaded_size *= num_blocks;

    // Add the results array
    size_t bt_offloaded_results_size = BT_OFFLOADED_RESULT_ELEMENTS(max_steps)
                                       * num_alignments;

    hipMemsetAsync(
        bt_offloaded_d,
        0,
        (bt_offloaded_size + bt_offloaded_results_size) * sizeof(wfa_backtrace_t),
        stream
    );
    CUDA_CHECK_ERR
}

size_t available_shared_mem_per_block (const affine_penalties_t penalties,
                                       const size_t max_steps,
                                       const int threads_per_block) {
    // TODO: Take band into account
    // TODO: Choose device from argument
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    CUDA_CHECK_ERR

    const size_t shared_mem_per_block = deviceProp.sharedMemPerBlock;
    const int max_occupancy = deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize;

    // Assume we get an occupancy of 32 warps per SM, with the limiting factor
    // being the registers used.
    const int max_active_warps_per_sm = min(32, max_occupancy);

    const int warps_per_block = threads_per_block / deviceProp.warpSize;
    const int blocks_per_sm = max_active_warps_per_sm / warps_per_block;
    const size_t usable_sh_mem_per_block = shared_mem_per_block / blocks_per_sm;

    LOG_DEBUG("Maximum usable shared memory per block: %.2fKiB. Maximum shared memory per block=%.2fKiB",
              (double)usable_sh_mem_per_block / (1<<10),
              (double)shared_mem_per_block / (1<<10))
    return usable_sh_mem_per_block;
}

size_t wf_data_buffer_size (const affine_penalties_t penalties,
                             const size_t max_steps) {
    const int max_wf_size = 2 * max_steps + 1;
    const int active_working_set = max(penalties.o+penalties.e, penalties.x) + 1;
    int offsets_elements = active_working_set * max_wf_size;
    offsets_elements = offsets_elements + (4 - (offsets_elements % 4));
    const int bt_elements = offsets_elements;
    size_t buffer_size =
                    // Offsets space
                    (offsets_elements * 3 * sizeof(wfa_offset_t))
                    // Backtraces vectors
                    + (bt_elements * 3 * sizeof(bt_vector_t))
                    // Backtraces pointers
                    + (bt_elements * 3 * sizeof(bt_prev_t));
    return buffer_size;
}

size_t wf_data_buffer_size_distance (const affine_penalties_t penalties,
                                     const size_t max_steps) {
    const int max_wf_size = 2 * max_steps + 1;
    const int active_working_set = max(penalties.o+penalties.e, penalties.x) + 1;
    int offsets_elements = active_working_set * max_wf_size;
    offsets_elements = offsets_elements + (4 - (offsets_elements % 4));
    size_t buffer_size = (offsets_elements * 3 * sizeof(wfa_offset_t));
    return buffer_size;
}

void allocate_wf_data_buffer_d (uint8_t** wf_data_buffer,
                                const size_t max_steps,
                                const affine_penalties_t penalties,
                                const size_t num_blocks) {

    // Create the active working set buffer on global memory
    size_t buffer_size = wf_data_buffer_size(penalties, max_steps);
    LOG_DEBUG("Working set size per block: %.2f KiB",
              (float)(buffer_size) / (1 << 10))
    buffer_size *= num_blocks;

    // Add a single int to be the global index of the next alignment in the pool
    buffer_size += sizeof(uint32_t);

    LOG_DEBUG("Allocating %.2f MiB to store working set data of %zu workers.",
              (float)(buffer_size) / (1 << 20), num_blocks)

    hipMalloc(wf_data_buffer, buffer_size);
    CUDA_CHECK_ERR;
}

void allocate_wf_data_buffer_distance_d (uint8_t** wf_data_buffer,
                                         const size_t max_steps,
                                         const affine_penalties_t penalties,
                                         const size_t num_blocks) {

    // Create the active working set buffer on global memory
    size_t buffer_size = wf_data_buffer_size_distance(penalties, max_steps);
    LOG_DEBUG("Working set size per block: %.2f KiB",
              (float)(buffer_size) / (1 << 10))
    buffer_size *= num_blocks;

    // Add a single int to be the global index of the next alignment in the pool
    buffer_size += sizeof(uint32_t);

    LOG_DEBUG("Allocating %.2f MiB to store working set data of %zu workers.",
              (float)(buffer_size) / (1 << 20), num_blocks)

    hipMalloc(wf_data_buffer, buffer_size);
    CUDA_CHECK_ERR;
}

void reset_wf_data_buffer_d (uint8_t* wf_data_buffer,
                             const size_t max_steps,
                             const affine_penalties_t penalties,
                             const size_t num_blocks,
                             hipStream_t stream) {

    // Create the active working set buffer on global memory
    size_t buffer_size = wf_data_buffer_size(penalties, max_steps);
    buffer_size *= num_blocks;
    buffer_size += sizeof(uint32_t);

    hipMemsetAsync(wf_data_buffer, 0, buffer_size, stream);
    CUDA_CHECK_ERR;
}

void reset_wf_data_buffer_distance_d (uint8_t* wf_data_buffer,
                                      const size_t max_steps,
                                      const affine_penalties_t penalties,
                                      const size_t num_blocks,
                                      hipStream_t stream) {

    // Create the active working set buffer on global memory
    size_t buffer_size = wf_data_buffer_size_distance(penalties, max_steps);
    buffer_size *= num_blocks;
    buffer_size += sizeof(uint32_t);

    hipMemsetAsync(wf_data_buffer, 0, buffer_size, stream);
    CUDA_CHECK_ERR;
}

void launch_alignments_async (const char* packed_sequences_buffer,
                              const sequence_pair_t* sequences_metadata,
                              const size_t num_alignments,
                              const affine_penalties_t penalties,
                              alignment_result_t* const results,
                              wfa_backtrace_t* const backtraces,
                              alignment_result_t *results_d,
                              wfa_backtrace_t* bt_offloaded_d,
                              uint8_t* const wf_data_buffer,
                              const int max_steps,
                              const int threads_per_block,
                              const int num_blocks,
                              int band,
                              hipStream_t stream) {
    // If band <= 0, make the alignment unbanded
    if (band <= 0) band = 2 * max_steps + 1;

    size_t bt_offloaded_size = BT_OFFLOADED_ELEMENTS(max_steps) * num_blocks;

    wfa_backtrace_t* bt_offloaded_results_d = bt_offloaded_d
                                              + bt_offloaded_size;

    // TODO: Reduction of penalties
    const int max_wf_size = 2 * max_steps + 1;
    const int active_working_set = max(penalties.o+penalties.e, penalties.x) + 1;
    int offsets_elements = active_working_set * max_wf_size;
    offsets_elements = offsets_elements + (4 - (offsets_elements % 4));

    size_t sh_mem_size = \
                    // Wavefronts structs space
                    + (active_working_set * sizeof(wfa_wavefront_t) * 3)
                    // Position of the last used element in the offloaded
                    // backtraces. It will be atomically increased.
                    + sizeof(int);

    size_t available_sh_mem_per_block = available_shared_mem_per_block(
                                            penalties,
                                            max_steps,
                                            threads_per_block) - sh_mem_size;
    // Using 100% of the shared memory available can give some problems, as the
    // driver sometimes take some sh memory space (?)
    available_sh_mem_per_block *= 0.95;

    const int max_sh_offsets_per_block = available_sh_mem_per_block / sizeof(wfa_offset_t);
    int max_sh_offsets_per_wf = max_sh_offsets_per_block / (active_working_set * 3);
    // Make it an odd number
    if ((max_sh_offsets_per_wf % 2) == 0) {
        max_sh_offsets_per_wf--;
    }

    // Make sure if fits in shared memory taking into account the wavefronts
    // metadata
    while ((max_sh_offsets_per_wf * sizeof(wfa_offset_t) * active_working_set * 3)
                > available_sh_mem_per_block) {
        max_sh_offsets_per_wf -= 2;
    }

    // Add offsets size to shared memory
    sh_mem_size += max_sh_offsets_per_wf * sizeof(wfa_offset_t) * active_working_set * 3;

    LOG_DEBUG("Each wavefront have %d offsets on shared memory", max_sh_offsets_per_wf)

    uint32_t* next_alignment_idx = (uint32_t*)(wf_data_buffer
                                           + wf_data_buffer_size(
                                               penalties,
                                               max_steps
                                           ) * num_blocks);
    dim3 gridSize(num_blocks);
    dim3 blockSize(threads_per_block);

    LOG_DEBUG("Launching %d blocks of %d threads with %.2fKiB of shared memory",
              gridSize.x, blockSize.x, (float(sh_mem_size) / (1 << 10)));

    LOG_DEBUG("Working with penalties: X=%d, O=%d, E=%d", penalties.x,
              penalties.o, penalties.e);

    alignment_kernel<<<gridSize, blockSize, sh_mem_size, stream>>>(
                                              packed_sequences_buffer,
                                              sequences_metadata,
                                              num_alignments,
                                              max_steps,
                                              wf_data_buffer,
                                              penalties,
                                              bt_offloaded_d,
                                              bt_offloaded_results_d,
                                              results_d,
                                              next_alignment_idx,
                                              max_sh_offsets_per_wf,
                                              band);
    CUDA_CHECK_ERR
}

void copyInResults (alignment_result_t* const results,
                    const alignment_result_t* const results_d,
                    wfa_backtrace_t* const backtraces,
                    const wfa_backtrace_t* const bt_offloaded_d,
                    const size_t num_alignments,
                    const int max_steps,
                    const int num_blocks,
                    hipStream_t stream) {
    const size_t bt_offloaded_results_size = BT_OFFLOADED_RESULT_ELEMENTS(max_steps)
                                             * num_alignments;
    const size_t bt_offloaded_size = BT_OFFLOADED_ELEMENTS(max_steps) * num_blocks;

    const wfa_backtrace_t* const bt_offloaded_results_d = bt_offloaded_d
                                              + bt_offloaded_size;

    hipMemcpyAsync(results, results_d, num_alignments * sizeof(alignment_result_t),
               hipMemcpyDeviceToHost, stream);
    CUDA_CHECK_ERR
    hipMemcpyAsync(backtraces, bt_offloaded_results_d,
               bt_offloaded_results_size * sizeof(wfa_backtrace_t),
               hipMemcpyDeviceToHost, stream);
    CUDA_CHECK_ERR
}

void launch_alignments_distance_async (const char* packed_sequences_buffer,
                                       const sequence_pair_t* sequences_metadata,
                                       const size_t num_alignments,
                                       const affine_penalties_t penalties,
                                       alignment_result_t* const results,
                                       alignment_result_t *results_d,
                                       uint8_t* const wf_data_buffer,
                                       const int max_steps,
                                       const int threads_per_block,
                                       const int num_blocks,
                                       int band,
                                       hipStream_t stream) {
    // If band <= 0, make the alignment unbanded
    if (band <= 0) band = 2 * max_steps + 1;

    const int max_wf_size = 2 * max_steps + 1;
    const int active_working_set = max(penalties.o+penalties.e, penalties.x) + 1;
    int offsets_elements = active_working_set * max_wf_size;
    offsets_elements = offsets_elements + (4 - (offsets_elements % 4));

    size_t sh_mem_size = (active_working_set * sizeof(wfa_wavefront_t) * 3);

    size_t available_sh_mem_per_block = available_shared_mem_per_block(
                                            penalties,
                                            max_steps,
                                            threads_per_block) - sh_mem_size;
    // Using 100% of the shared memory available can give some problems, as the
    // driver sometimes take some sh memory space (?)
    available_sh_mem_per_block *= 0.95;

    const int max_sh_offsets_per_block = available_sh_mem_per_block / sizeof(wfa_offset_t);
    int max_sh_offsets_per_wf = max_sh_offsets_per_block / (active_working_set * 3);
    // Make it an odd number
    if ((max_sh_offsets_per_wf % 2) == 0) {
        max_sh_offsets_per_wf--;
    }

    // Make sure if fits in shared memory taking into account the wavefronts
    // metadata
    while ((max_sh_offsets_per_wf * sizeof(wfa_offset_t) * active_working_set * 3)
                > available_sh_mem_per_block) {
        max_sh_offsets_per_wf -= 2;
    }

    // Add offsets size to shared memory
    sh_mem_size += max_sh_offsets_per_wf * sizeof(wfa_offset_t) * active_working_set * 3;

    LOG_DEBUG("Each wavefront have %d offsets on shared memory", max_sh_offsets_per_wf)

    uint32_t* next_alignment_idx = (uint32_t*)(wf_data_buffer
                                           + wf_data_buffer_size_distance(
                                               penalties,
                                               max_steps
                                           ) * num_blocks);
    dim3 gridSize(num_blocks);
    dim3 blockSize(threads_per_block);

    LOG_DEBUG("Launching %d blocks of %d threads with %.2fKiB of shared memory",
              gridSize.x, blockSize.x, (float(sh_mem_size) / (1 << 10)));

    LOG_DEBUG("Working with penalties: X=%d, O=%d, E=%d", penalties.x,
              penalties.o, penalties.e);

    distance_kernel<<<gridSize, blockSize, sh_mem_size, stream>>>(
                                              packed_sequences_buffer,
                                              sequences_metadata,
                                              num_alignments,
                                              max_steps,
                                              wf_data_buffer,
                                              penalties,
                                              results_d,
                                              next_alignment_idx,
                                              max_sh_offsets_per_wf,
                                              band);
    CUDA_CHECK_ERR
}

void copyInResults_distance (alignment_result_t* const results,
                             const alignment_result_t* const results_d,
                             const size_t num_alignments,
                             hipStream_t stream) {

    hipMemcpyAsync(results, results_d, num_alignments * sizeof(alignment_result_t),
               hipMemcpyDeviceToHost, stream);
    CUDA_CHECK_ERR
}
