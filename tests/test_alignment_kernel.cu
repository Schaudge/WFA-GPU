/*
 * Copyright (c) 2021 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <stdint.h>

#include "utils/logger.h"
#include "utils/sequences.h"
#include "sequence_packing.cuh"
#include "sequence_alignment.cuh"
#include "wfa_types.h"
#include "affine_penalties.h"
#include "alignment_results.h"
#include "tests/test.h"

#define EWAVEFRONT_V(k,offset) ((offset)-(k))
#define EWAVEFRONT_H(k,offset) (offset)

SET_TEST_NAME("ALIGNMENT KERNEL")

wfa_offset_t extend_wavefront (
        const wfa_offset_t offset_val,
        const int curr_k,
        const char* const pattern,
        const int pattern_length,
        const char* const text,
        const int text_length) {
    // Parameters
    int v = EWAVEFRONT_V(curr_k, offset_val);
    int h = EWAVEFRONT_H(curr_k, offset_val);
    wfa_offset_t acc = 0;
    while (v<pattern_length && h<text_length && pattern[v++]==text[h++]) {
      acc++;
    }
    return acc;
}

char* recover_cigar (const char* text,
                     const char* pattern,
                     const size_t tlen,
                     const size_t plen,
                     wfa_backtrace_t backtrace) {

    char* cigar_ascii = (char*)calloc(tlen + plen, 1);
    char* cigar_ptr = cigar_ascii;

    int steps = 16 - (__builtin_clz(backtrace) / 2);

    int k=0;
    wfa_offset_t offset = 0;
    for (int d=0; d<steps; d++) {
        wfa_offset_t acc = extend_wavefront(offset, k, pattern, plen, text, tlen);
        for (int j=0; j<acc; j++) {
            *cigar_ptr = 'M';
            cigar_ptr++;
        }

        offset += acc;

        affine_op_t op = (affine_op_t)((backtrace >> ((steps - d - 1) * 2)) & 3);

        switch (op) {
            // k + 1
            case OP_DEL:
                *cigar_ptr = 'D';
                k--;
                break;
            // k
            case OP_SUB:
                *cigar_ptr = 'X';
                offset++;
                break;
            // k - 1
            case OP_INS:
                *cigar_ptr = 'I';
                k++;
                offset++;
                break;
        }
        cigar_ptr++;
    }

    // Last exension
    wfa_offset_t acc = extend_wavefront(offset, k, pattern, plen, text, tlen);
    for (int j=0; j<acc; j++) {
        *cigar_ptr = 'M';
        cigar_ptr++;
    }


    return cigar_ascii;
}

void test_one_alignment() {
    // One sequence test
    size_t seq_buf_size = 32;
    char* sequence_unpacked = (char*)calloc(seq_buf_size, 1);
    sequence_pair_t* sequence_metadata = (sequence_pair_t*)calloc(1, sizeof(sequence_pair_t));
    if (!sequence_unpacked || !sequence_metadata) {
        LOG_ERROR("Can not allocate memory");
        exit(-1);
    }

    sequence_metadata[0].pattern_offset = 0;
    sequence_metadata[0].pattern_len = 7;
    strcpy(sequence_unpacked, "GATTACA");

    sequence_metadata[0].text_offset = 12;
    sequence_metadata[0].text_len = 5;
    strcpy(sequence_unpacked + sequence_metadata[0].text_offset, "GAATA");
    size_t num_alignments = 1;

    char* d_seq_buf_unpacked = NULL;
    char* d_seq_buf_packed = NULL;
    size_t d_seq_buf_packed_size = 0;
    sequence_pair_t* d_seq_metadata = NULL;

    prepare_pack_sequences_gpu_async(
        sequence_unpacked,
        seq_buf_size,
        sequence_metadata,
        num_alignments,
        &d_seq_buf_unpacked,
        &d_seq_buf_packed,
        &d_seq_buf_packed_size,
        &d_seq_metadata,
        0
    );

    pack_sequences_gpu_async(
        d_seq_buf_unpacked,
        d_seq_buf_packed,
        seq_buf_size,
        d_seq_buf_packed_size,
        d_seq_metadata,
        num_alignments,
        0
    );


    affine_penalties_t penalties = {.x = 2, .o = 3, .e = 1};
    //// Only one sequence in this test
    alignment_result_t results = {0};

    launch_alignments_async(
        d_seq_buf_packed,
        d_seq_metadata,
        num_alignments,
        penalties,
        &results
    );

    hipDeviceSynchronize();

    TEST_ASSERT(results.distance == 7)

    penalties = {.x = 1, .o = 0, .e = 1};

    launch_alignments_async(
        d_seq_buf_packed,
        d_seq_metadata,
        num_alignments,
        penalties,
        &results
    );

    hipDeviceSynchronize();

    TEST_ASSERT(results.distance == 3)

    hipFree(d_seq_buf_unpacked);
    hipFree(d_seq_buf_packed);
    hipFree(d_seq_metadata);
    free(sequence_unpacked);
    free(sequence_metadata);

    hipDeviceSynchronize();
}

void test_multiple_alignments_affine () {
    // >TGTGAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG
    // <TGTAAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG
    // >ACGGGCGTGCATCACAACCCGTGATGATCGCCATAGAGCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGTCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA
    // <ACGGGCGTGCATCACAACCCGGATGATCGCCATAGAGCCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGATCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA
    // >ATACCCCCGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGATGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG
    // <ATCCCACGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGTGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG

    size_t seq_buf_size = 1024;
    char* sequence_unpacked = (char*)calloc(seq_buf_size, 1);
    sequence_pair_t* sequence_metadata = (sequence_pair_t*)calloc(3, sizeof(sequence_pair_t));
    if (!sequence_unpacked || !sequence_metadata) {
        LOG_ERROR("Can not allocate memory");
        exit(-1);
    }

    sequence_metadata[0].pattern_offset = 0;
    sequence_metadata[0].pattern_len = 150;
    strcpy(sequence_unpacked, "TGTGAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG");

    sequence_metadata[0].text_offset = 152;
    sequence_metadata[0].text_len = 151;
    strcpy(sequence_unpacked + sequence_metadata[0].text_offset, "TGTAAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG");


    sequence_metadata[1].pattern_offset = 308;
    sequence_metadata[1].pattern_len = 150;
    strcpy(sequence_unpacked + sequence_metadata[1].pattern_offset, "ACGGGCGTGCATCACAACCCGTGATGATCGCCATAGAGCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGTCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA");

    sequence_metadata[1].text_offset = 460;
    sequence_metadata[1].text_len = 151;
    strcpy(sequence_unpacked + sequence_metadata[1].text_offset, "ACGGGCGTGCATCACAACCCGGATGATCGCCATAGAGCCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGATCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA");

    sequence_metadata[2].pattern_offset = 616;
    sequence_metadata[2].pattern_len = 150;
    strcpy(sequence_unpacked + sequence_metadata[2].pattern_offset, "ATACCCCCGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGATGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG");

    sequence_metadata[2].text_offset = 768;
    sequence_metadata[2].text_len = 148;
    strcpy(sequence_unpacked + sequence_metadata[2].text_offset, "ATCCCACGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGTGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG");
    size_t num_alignments = 3;

    char* d_seq_buf_unpacked = NULL;
    char* d_seq_buf_packed = NULL;
    size_t d_seq_buf_packed_size = 0;
    sequence_pair_t* d_seq_metadata = NULL;

    prepare_pack_sequences_gpu_async(
        sequence_unpacked,
        seq_buf_size,
        sequence_metadata,
        num_alignments,
        &d_seq_buf_unpacked,
        &d_seq_buf_packed,
        &d_seq_buf_packed_size,
        &d_seq_metadata,
        0
    );

    pack_sequences_gpu_async(
        d_seq_buf_unpacked,
        d_seq_buf_packed,
        seq_buf_size,
        d_seq_buf_packed_size,
        d_seq_metadata,
        num_alignments,
        0
    );

    hipDeviceSynchronize();

    affine_penalties_t penalties = {.x = 2, .o = 3, .e = 1};
    alignment_result_t* results = (alignment_result_t*)calloc(num_alignments,
                                                              sizeof(alignment_result_t));

    launch_alignments_async(
        d_seq_buf_packed,
        d_seq_metadata,
        num_alignments,
        penalties,
        results
    );

    hipDeviceSynchronize();

    const int correct_results[3] = {6, 12, 10};
    const char* correct_cigars[3] = {
        "MMMXMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMIMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMM",
        "MMMMMMMMMMMMMMMMMMMMMDMMMMMMMMMMMMMMMMMIMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMIMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMM",
        "MMDMMMXMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMDMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMM"
    };

    for (int i=0; i<num_alignments; i++) {
        // TODO
        char* text = &sequence_unpacked[sequence_metadata[i].text_offset];
        char* pattern = &sequence_unpacked[sequence_metadata[i].pattern_offset];
        size_t tlen = sequence_metadata[i].text_len;
        size_t plen = sequence_metadata[i].pattern_len;
        int distance = results[i].distance;
        char* cigar = recover_cigar(text, pattern, tlen,
                                    plen,results[i].backtrace);

        TEST_ASSERT(distance == correct_results[i])
        TEST_ASSERT(!strcmp(cigar, correct_cigars[i]))
    }

    hipFree(d_seq_buf_unpacked);
    hipFree(d_seq_buf_packed);
    hipFree(d_seq_metadata);
    free(sequence_unpacked);
    free(sequence_metadata);
    free(results);

    hipDeviceSynchronize();
}

void test_multiple_alignments_edit () {
    // >TGTGAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG
    // <TGTAAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG
    // >ACGGGCGTGCATCACAACCCGTGATGATCGCCATAGAGCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGTCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA
    // <ACGGGCGTGCATCACAACCCGGATGATCGCCATAGAGCCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGATCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA
    // >ATACCCCCGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGATGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG
    // <ATCCCACGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGTGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG

    size_t seq_buf_size = 1024;
    char* sequence_unpacked = (char*)calloc(seq_buf_size, 1);
    sequence_pair_t* sequence_metadata = (sequence_pair_t*)calloc(3, sizeof(sequence_pair_t));
    if (!sequence_unpacked || !sequence_metadata) {
        LOG_ERROR("Can not allocate memory");
        exit(-1);
    }

    sequence_metadata[0].pattern_offset = 0;
    sequence_metadata[0].pattern_len = 150;
    strcpy(sequence_unpacked, "TGTGAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG");

    sequence_metadata[0].text_offset = 152;
    sequence_metadata[0].text_len = 151;
    strcpy(sequence_unpacked + sequence_metadata[0].text_offset, "TGTAAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG");


    sequence_metadata[1].pattern_offset = 308;
    sequence_metadata[1].pattern_len = 150;
    strcpy(sequence_unpacked + sequence_metadata[1].pattern_offset, "ACGGGCGTGCATCACAACCCGTGATGATCGCCATAGAGCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGTCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA");

    sequence_metadata[1].text_offset = 460;
    sequence_metadata[1].text_len = 151;
    strcpy(sequence_unpacked + sequence_metadata[1].text_offset, "ACGGGCGTGCATCACAACCCGGATGATCGCCATAGAGCCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGATCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA");

    sequence_metadata[2].pattern_offset = 616;
    sequence_metadata[2].pattern_len = 150;
    strcpy(sequence_unpacked + sequence_metadata[2].pattern_offset, "ATACCCCCGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGATGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG");

    sequence_metadata[2].text_offset = 768;
    sequence_metadata[2].text_len = 148;
    strcpy(sequence_unpacked + sequence_metadata[2].text_offset, "ATCCCACGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGTGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG");
    size_t num_alignments = 3;

    char* d_seq_buf_unpacked = NULL;
    char* d_seq_buf_packed = NULL;
    size_t d_seq_buf_packed_size = 0;
    sequence_pair_t* d_seq_metadata = NULL;

    prepare_pack_sequences_gpu_async(
        sequence_unpacked,
        seq_buf_size,
        sequence_metadata,
        num_alignments,
        &d_seq_buf_unpacked,
        &d_seq_buf_packed,
        &d_seq_buf_packed_size,
        &d_seq_metadata,
        0
    );

    pack_sequences_gpu_async(
        d_seq_buf_unpacked,
        d_seq_buf_packed,
        seq_buf_size,
        d_seq_buf_packed_size,
        d_seq_metadata,
        num_alignments,
        0
    );

    hipDeviceSynchronize();

    //affine_penalties_t penalties = {.x = 2, .o = 3, .e = 1};
    affine_penalties_t penalties = {.x = 1, .o = 0, .e = 1};
    alignment_result_t* results = (alignment_result_t*)calloc(num_alignments,
                                                              sizeof(alignment_result_t));

    launch_alignments_async(
        d_seq_buf_packed,
        d_seq_metadata,
        num_alignments,
        penalties,
        results
    );

    hipDeviceSynchronize();

    const int correct_results[3] = {2, 3, 3};
    const char* correct_cigars[3] = {
        "MMMXMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMIMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMM",
        "MMMMMMMMMMMMMMMMMMMMMDMMMMMMMMMMMMMMMMMIMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMIMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMM",
        "MMDMMMXMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMDMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMMM"
    };

    for (int i=0; i<num_alignments; i++) {
        // TODO
        char* text = &sequence_unpacked[sequence_metadata[i].text_offset];
        char* pattern = &sequence_unpacked[sequence_metadata[i].pattern_offset];
        size_t tlen = sequence_metadata[i].text_len;
        size_t plen = sequence_metadata[i].pattern_len;
        int distance = results[i].distance;
        char* cigar = recover_cigar(text, pattern, tlen,
                                    plen,results[i].backtrace);

        TEST_ASSERT(distance == correct_results[i])
        TEST_ASSERT(!strcmp(cigar, correct_cigars[i]))
    }

    hipFree(d_seq_buf_unpacked);
    hipFree(d_seq_buf_packed);
    hipFree(d_seq_metadata);
    free(sequence_unpacked);
    free(sequence_metadata);
    free(results);

    hipDeviceSynchronize();
}

int main () {

    test_one_alignment();
    test_multiple_alignments_edit();
    test_multiple_alignments_affine();

    TEST_OK
    return 0;
}
