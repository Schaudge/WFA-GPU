/*
 * Copyright (c) 2021 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <stdint.h>

#include "utils/logger.h"
#include "utils/sequences.h"
#include "sequence_packing.cuh"
#include "sequence_alignment.cuh"
#include "affine_penalties.h"
#include "alignment_results.h"
#include "tests/test.h"

SET_TEST_NAME("ALIGNMENT KERNEL")

void test_one_alignment() {
    // One sequence test
    size_t seq_buf_size = 32;
    char* sequence_unpacked = (char*)calloc(seq_buf_size, 1);
    sequence_pair_t* sequence_metadata = (sequence_pair_t*)calloc(1, sizeof(sequence_pair_t));
    if (!sequence_unpacked || !sequence_metadata) {
        LOG_ERROR("Can not allocate memory");
        exit(-1);
    }

    sequence_metadata[0].pattern_offset = 0;
    sequence_metadata[0].pattern_len = 7;
    strcpy(sequence_unpacked, "GATTACA");

    sequence_metadata[0].text_offset = 12;
    sequence_metadata[0].text_len = 5;
    strcpy(sequence_unpacked + sequence_metadata[0].text_offset, "GAATA");
    size_t num_alignments = 1;

    char* d_seq_buf_unpacked = NULL;
    char* d_seq_buf_packed = NULL;
    size_t d_seq_buf_packed_size = 0;
    sequence_pair_t* d_seq_metadata = NULL;

    prepare_pack_sequences_gpu_async(
        sequence_unpacked,
        seq_buf_size,
        sequence_metadata,
        num_alignments,
        &d_seq_buf_unpacked,
        &d_seq_buf_packed,
        &d_seq_buf_packed_size,
        &d_seq_metadata,
        0
    );

    pack_sequences_gpu_async(
        d_seq_buf_unpacked,
        d_seq_buf_packed,
        seq_buf_size,
        d_seq_buf_packed_size,
        d_seq_metadata,
        num_alignments,
        0
    );


    affine_penalties_t penalties = {.x = 2, .o = 3, .e = 1};
    //// Only one sequence in this test
    alignment_result_t results = {0};

    launch_alignments_async(
        d_seq_buf_packed,
        d_seq_metadata,
        num_alignments,
        penalties,
        &results
    );

    hipDeviceSynchronize();

    TEST_ASSERT(results.distance == 7)

    penalties = {.x = 1, .o = 0, .e = 1};

    launch_alignments_async(
        d_seq_buf_packed,
        d_seq_metadata,
        num_alignments,
        penalties,
        &results
    );

    hipDeviceSynchronize();

    TEST_ASSERT(results.distance == 3)

    hipFree(d_seq_buf_unpacked);
    hipFree(d_seq_buf_packed);
    hipFree(d_seq_metadata);
    free(sequence_unpacked);
    free(sequence_metadata);

    hipDeviceSynchronize();
}

void test_multiple_alignments () {
    // >TGTGAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG
    // <TGTAAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG
    // >ACGGGCGTGCATCACAACCCGTGATGATCGCCATAGAGCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGTCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA
    // <ACGGGCGTGCATCACAACCCGGATGATCGCCATAGAGCCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGATCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA
    // >ATACCCCCGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGATGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG
    // <ATCCCACGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGTGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG

    size_t seq_buf_size = 1024;
    char* sequence_unpacked = (char*)calloc(seq_buf_size, 1);
    sequence_pair_t* sequence_metadata = (sequence_pair_t*)calloc(3, sizeof(sequence_pair_t));
    if (!sequence_unpacked || !sequence_metadata) {
        LOG_ERROR("Can not allocate memory");
        exit(-1);
    }

    sequence_metadata[0].pattern_offset = 0;
    sequence_metadata[0].pattern_len = 150;
    strcpy(sequence_unpacked, "TGTGAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG");

    sequence_metadata[0].text_offset = 152;
    sequence_metadata[0].text_len = 151;
    strcpy(sequence_unpacked + sequence_metadata[0].text_offset, "TGTAAAGTAATGGACGTTCTATTGGTTAAGAAATGCACCAGCTACAGCCAAACTATGAGTCATCCTTTTCCATGTTAAGCCTGGTTCCTAAACACTTCGTGAAGGACGAAACTTATGCACGCGTCTGCCCAACAGAAATCCTTCGTAACCG");


    sequence_metadata[1].pattern_offset = 308;
    sequence_metadata[1].pattern_len = 150;
    strcpy(sequence_unpacked + sequence_metadata[1].pattern_offset, "ACGGGCGTGCATCACAACCCGTGATGATCGCCATAGAGCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGTCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA");

    sequence_metadata[1].text_offset = 460;
    sequence_metadata[1].text_len = 151;
    strcpy(sequence_unpacked + sequence_metadata[1].text_offset, "ACGGGCGTGCATCACAACCCGGATGATCGCCATAGAGCCGAGGGGTGGATATGGAGACCGTGTTGACGGTCTCACATATATTTGGTCTAGCACCTTCCGACATGACTTCGATCCTAATCTTACTCGTCAAAACAAAACAATGACAAGATAA");

    sequence_metadata[2].pattern_offset = 616;
    sequence_metadata[2].pattern_len = 150;
    strcpy(sequence_unpacked + sequence_metadata[2].pattern_offset, "ATACCCCCGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGATGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG");

    sequence_metadata[2].text_offset = 768;
    sequence_metadata[2].text_len = 148;
    strcpy(sequence_unpacked + sequence_metadata[2].text_offset, "ATCCCACGTCTTATCATACGACCCTAATGCACGCGTTAGGGCGGCTTAAATCCCTCCTATCCCTGATGCCATTTGATGTGAAACTCGTGGCTAAGAAACGCCCAACTGGTCGTCTTTGTCCACCCTGGAAACGCGGGCACCCTCTTAG");
    size_t num_alignments = 3;

    char* d_seq_buf_unpacked = NULL;
    char* d_seq_buf_packed = NULL;
    size_t d_seq_buf_packed_size = 0;
    sequence_pair_t* d_seq_metadata = NULL;

    prepare_pack_sequences_gpu_async(
        sequence_unpacked,
        seq_buf_size,
        sequence_metadata,
        num_alignments,
        &d_seq_buf_unpacked,
        &d_seq_buf_packed,
        &d_seq_buf_packed_size,
        &d_seq_metadata,
        0
    );

    pack_sequences_gpu_async(
        d_seq_buf_unpacked,
        d_seq_buf_packed,
        seq_buf_size,
        d_seq_buf_packed_size,
        d_seq_metadata,
        num_alignments,
        0
    );

    hipDeviceSynchronize();

    //affine_penalties_t penalties = {.x = 2, .o = 3, .e = 1};
    affine_penalties_t penalties = {.x = 1, .o = 0, .e = 1};
    alignment_result_t* results = (alignment_result_t*)calloc(num_alignments,
                                                              sizeof(alignment_result_t));

    launch_alignments_async(
        d_seq_buf_packed,
        d_seq_metadata,
        num_alignments,
        penalties,
        results
    );

    hipDeviceSynchronize();

    for (int i=0; i<num_alignments; i++) {
        printf("Alignment: %d, distance: %d\n", i, results[i].distance);
    }

    //TEST_ASSERT(results.distance == 7)

    //penalties = {.x = 1, .o = 0, .e = 1};

    //launch_alignments_async(
    //    d_seq_buf_packed,
    //    d_seq_metadata,
    //    num_alignments,
    //    penalties,
    //    &results
    //);

    //hipDeviceSynchronize();

    //TEST_ASSERT(results.distance == 3)

    hipFree(d_seq_buf_unpacked);
    hipFree(d_seq_buf_packed);
    hipFree(d_seq_metadata);
    free(sequence_unpacked);
    free(sequence_metadata);
    free(results);

    hipDeviceSynchronize();
}

int main () {

    test_one_alignment();
    test_multiple_alignments();

    TEST_OK
    return 0;
}
