/*
 * Copyright (c) 2021 Quim Aguado
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal in
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
 * the Software, and to permit persons to whom the Software is furnished to do so,
 * subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
 * FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
 * COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
 * IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
 * CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include <string.h>
#include "device_query.cuh"
#include "cuda_utils.cuh"

void get_num_cuda_devices(int* n) {
    hipGetDeviceCount(n);
    CUDA_CHECK_ERR
}

char* get_cuda_dev_name(int dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    CUDA_CHECK_ERR
    // TODO: Check strdup err
    return strdup(deviceProp.name);
}

void get_cuda_capability(int dev, int* major, int* minor) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    CUDA_CHECK_ERR

    *major = deviceProp.major;
    *minor = deviceProp.minor;
}
